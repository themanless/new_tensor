#include "hip/hip_runtime.h"
#include "twoalg.h"

void tensor2maxtr(float *T,float *Ac,int a,int b,int c){ 
	float *T1 = new float[a*b*c];
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int t = 0;t<b;t++){
				T1[t*a*c+j*c+i] = T[i*a*b+j*b+t];   
			}
		}
	}                  //a*b*c变为a*c*b
	for(int i = 0;i<b;i++){
		for(int j = 0;j<a;j++){
			for(int k = 0;k<c;k++){
				T[i*a*c+k*a+j] = T1[i*a*c+j*c+k];
			}
			
		}
	}                 //a c转置一下 c*a*b
	for(int i = 0;i<b;i++){
		for(int j = 0;j<a*c;j++){
			T1[j*b+i] = T[i*a*c+j];    //T为按矩阵的行读写，再将矩阵转置 ，T1已经变成按行存储ac*b
		}
	}
	
	for(int i = 0;i<a*c;i++){
		
		for(int j = 0;j<b;j++){

			Ac[i*b*c+j] = T1[i*b+j];   //将原矩阵赋值进入Ac
				
		}
	}
	
	for(int k = 1;k<c;k++){
		for(int i = 0;i<a*c;i++){
			for(int j = 0;j<b;j++){
				Ac[i*b*c+k*b+j] = T1[((i+(c-k)*a)%(a*c))*b+j];
			}
		}            			//矩阵循环后赋值进入AC中
	}

}



void Msvd(float *A,float *U,float *S,float *V,int m,int n){   //实现矩阵的svd，A的大小为m*n
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream = NULL;
	hipsolverGesvdjInfo_t gesvdj_params = NULL;   //创建句柄	
	
	const int lda = m; //矩阵A的主维度
	//显存端分配空间
	
	float *d_A = NULL; /* device copy of A */
	float *d_S = NULL; /* singular values */
	float *d_U = NULL; /* left singular vectors */
	float *d_V = NULL; /* right singular vectors */
	int *d_info = NULL; /* error info */
	int lwork = 0;
	/* size of workspace */
	float *d_work = NULL; /* devie workspace for gesvdj */
	int info = 0;	/* host copy of error info */

	/* configuration of gesvdj */
	const double tol = 1.e-7;
	const int max_sweeps = 15;
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
	const int econ = 0 ; /* econ = 1 for economy size */
	/* numerical results of gesvdj*/
	
	hipsolverDnCreate(&cusolverH);
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	hipsolverSetStream(cusolverH, stream);
	hipsolverDnCreateGesvdjInfo(&gesvdj_params);
	hipsolverDnXgesvdjSetTolerance(
		gesvdj_params,
		tol);

	hipsolverDnXgesvdjSetMaxSweeps(
		gesvdj_params,
		max_sweeps);
	hipMalloc((void**)&d_A,sizeof(float)*lda*n);
	hipMalloc((void**)&d_S,sizeof(float)*n);
	hipMalloc((void**)&d_U,sizeof(float)*lda*m);
	hipMalloc((void**)&d_V,sizeof(float)*n*n);
	hipMalloc((void**)&d_info,sizeof(float));

	hipMemcpy(d_A, A, sizeof(float)*lda*n,hipMemcpyHostToDevice); //A传到GPU端
	
	hipsolverDnSgesvdj_bufferSize(
		cusolverH,
		jobz, 	/* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
			/* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singularvectors */
		econ,    /* econ = 1 for economy size */
		m,    /* nubmer of rows of A, 0 <= m */
		n,   /* number of columns of A, 0 <= n */
		d_A,  /* m-by-n */
		lda,  /* leading dimension of A */
		d_S,  /* min(m,n) */
			/* the singular values in descending order */
		d_U,   /* m-by-m if econ = 0 */
			/* m-by-min(m,n) if econ = 1 */
		lda,    /* leading dimension of U, ldu >= max(1,m) */
		d_V,   /* n-by-n if econ = 0 */
			/* n-by-min(m,n) if econ = 1 */
		n,   	/* leading dimension of V, ldv >= max(1,n) */
		&lwork,
		gesvdj_params);

	hipMalloc((void**)&d_work , sizeof(float)*lwork);

	hipsolverDnSgesvdj(
	cusolverH,
		jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
			/* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singularvectors */
		econ, 	/* econ = 1 for economy size */
		m, 	/* nubmer of rows of A, 0 <= m */
		n,	/* number of columns of A, 0 <= n */
		d_A,	/* m-by-n */
		lda,	/* leading dimension of A */
		d_S,	/* min(m,n) */
			/* the singular values in descending order */
		d_U,
			/* m-by-m if econ = 0 */
			/* m-by-min(m,n) if econ = 1 */
		lda,
			/* leading dimension of U, ldu >= max(1,m) */
		d_V,
			/* n-by-n if econ = 0 */
			/* n-by-min(m,n) if econ = 1 */
		n,
			/* leading dimension of V, ldv >= max(1,n) */
		d_work,
		lwork,
		d_info,
		gesvdj_params);

	hipDeviceSynchronize();

	hipMemcpy(U, d_U, sizeof(float)*lda*m,hipMemcpyDeviceToHost);
	hipMemcpy(V, d_V, sizeof(float)*n*n,hipMemcpyDeviceToHost);
	hipMemcpy(S, d_S, sizeof(float)*n,hipMemcpyDeviceToHost);
	hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	if ( 0 == info ){
	printf("gesvdj converges \n");
	}else if ( 0 > info ){
	printf("%d-th parameter is wrong \n", -info);
	exit(1);
	}else{
	printf("WARNING: info = %d : gesvdj does not converge \n", info );
	}
	printf("=====\n");

	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_V);
	hipFree(d_info);
	hipFree(d_work);
	
	hipsolverDnDestroy(cusolverH);
	hipStreamDestroy(stream);
	hipsolverDnDestroyGesvdjInfo(gesvdj_params);
	hipDeviceReset();

}





