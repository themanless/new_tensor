#include "hip/hip_runtime.h"
#include "twoalg.h"

void tensor2maxtr(float *T,float *Ac,int a,int b,int c){ 
	float *T1 = new float[a*b*c];
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int t = 0;t<b;t++){
				T1[t*a*c+j*c+i] = T[i*a*b+j*b+t];   
			}
		}
	}                  //a*b*c变为a*c*b
	for(int i = 0;i<b;i++){
		for(int j = 0;j<a;j++){
			for(int k = 0;k<c;k++){
				T[i*a*c+k*a+j] = T1[i*a*c+j*c+k];
			}
			
		}
	}                 //a c转置一下 c*a*b
	for(int i = 0;i<b;i++){
		for(int j = 0;j<a*c;j++){
			T1[j*b+i] = T[i*a*c+j];    //T为按矩阵的行读写，再将矩阵转置 ，T1已经变成按行存储ac*b
		}
	}
	
	for(int i = 0;i<a*c;i++){
		
		for(int j = 0;j<b;j++){

			Ac[i*b*c+j] = T1[i*b+j];   //将原矩阵赋值进入Ac
				
		}
	}
	
	for(int k = 1;k<c;k++){
		for(int i = 0;i<a*c;i++){
			for(int j = 0;j<b;j++){
				Ac[i*b*c+k*b+j] = T1[((i+(c-k)*a)%(a*c))*b+j];
			}
		}            			//矩阵循环后赋值进入AC中
	}

}



void Msvd(float *A,float *U,float *S,float *V,int m,int n){   //实现矩阵的svd，A的大小为m*n
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream = NULL;
	hipsolverGesvdjInfo_t gesvdj_params = NULL;   //创建句柄	
	
	const int lda = m; //矩阵A的主维度
	//显存端分配空间
	
	float *d_A = NULL; /* device copy of A */
	float *d_S = NULL; /* singular values */
	float *d_U = NULL; /* left singular vectors */
	float *d_V = NULL; /* right singular vectors */
	int *d_info = NULL; /* error info */
	int lwork = 0;
	/* size of workspace */
	float *d_work = NULL; /* devie workspace for gesvdj */
	int info = 0;	/* host copy of error info */

	/* configuration of gesvdj */
	const double tol = 1.e-7;
	const int max_sweeps = 15;
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
	const int econ = 0 ; /* econ = 1 for economy size */
	/* numerical results of gesvdj*/
	
	hipsolverDnCreate(&cusolverH);
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	hipsolverSetStream(cusolverH, stream);
	hipsolverDnCreateGesvdjInfo(&gesvdj_params);
	hipsolverDnXgesvdjSetTolerance(
		gesvdj_params,
		tol);

	hipsolverDnXgesvdjSetMaxSweeps(
		gesvdj_params,
		max_sweeps);
	hipMalloc((void**)&d_A,sizeof(float)*lda*n);
	hipMalloc((void**)&d_S,sizeof(float)*n);
	hipMalloc((void**)&d_U,sizeof(float)*lda*m);
	hipMalloc((void**)&d_V,sizeof(float)*n*n);
	hipMalloc((void**)&d_info,sizeof(float));

	hipMemcpy(d_A, A, sizeof(float)*lda*n,hipMemcpyHostToDevice); //A传到GPU端
	
	hipsolverDnSgesvdj_bufferSize(
		cusolverH,
		jobz, 	/* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
			/* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singularvectors */
		econ,    /* econ = 1 for economy size */
		m,    /* nubmer of rows of A, 0 <= m */
		n,   /* number of columns of A, 0 <= n */
		d_A,  /* m-by-n */
		lda,  /* leading dimension of A */
		d_S,  /* min(m,n) */
			/* the singular values in descending order */
		d_U,   /* m-by-m if econ = 0 */
			/* m-by-min(m,n) if econ = 1 */
		lda,    /* leading dimension of U, ldu >= max(1,m) */
		d_V,   /* n-by-n if econ = 0 */
			/* n-by-min(m,n) if econ = 1 */
		n,   	/* leading dimension of V, ldv >= max(1,n) */
		&lwork,
		gesvdj_params);

	hipMalloc((void**)&d_work , sizeof(float)*lwork);

	hipsolverDnSgesvdj(
		cusolverH,
		jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
			/* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singularvectors */
		econ, 	/* econ = 1 for economy size */
		m, 	/* nubmer of rows of A, 0 <= m */
		n,	/* number of columns of A, 0 <= n */
		d_A,	/* m-by-n */
		lda,	/* leading dimension of A */
		d_S,	/* min(m,n) */
			/* the singular values in descending order */
		d_U,
			/* m-by-m if econ = 0 */
			/* m-by-min(m,n) if econ = 1 */
		lda,
			/* leading dimension of U, ldu >= max(1,m) */
		d_V,
			/* n-by-n if econ = 0 */
			/* n-by-min(m,n) if econ = 1 */
		n,
			/* leading dimension of V, ldv >= max(1,n) */
		d_work,
		lwork,
		d_info,
		gesvdj_params);

	hipDeviceSynchronize();

	hipMemcpy(U, d_U, sizeof(float)*lda*m,hipMemcpyDeviceToHost);
	hipMemcpy(V, d_V, sizeof(float)*n*n,hipMemcpyDeviceToHost);
	hipMemcpy(S, d_S, sizeof(float)*n,hipMemcpyDeviceToHost);
	hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	if ( 0 == info ){
	printf("gesvdj converges \n");
	}else if ( 0 > info ){
	printf("%d-th parameter is wrong \n", -info);
	exit(1);
	}else{
	printf("WARNING: info = %d : gesvdj does not converge \n", info );
	}
	printf("=====\n");

	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_V);
	hipFree(d_info);
	hipFree(d_work);
	
	hipsolverDnDestroy(cusolverH);
	hipStreamDestroy(stream);
	hipsolverDnDestroyGesvdjInfo(gesvdj_params);
	hipDeviceReset();

}

void Mfsvd(hipfftComplex *A,hipfftComplex *U,float *S,hipfftComplex *V,int m,int n){   //实现矩阵的svd，A的大小为m*n
	hipsolverHandle_t cusolverH = NULL;   //创建句柄	返回的是Ｖ的转置　按列存储的
	
	const int lda = m; //矩阵A的主维度
	//显存端分配空间
	
	hipfftComplex *d_A = NULL; /* device copy of A */
	float *d_S = NULL; /* singular values */
	hipfftComplex *d_U = NULL; /* left singular vectors */
	hipfftComplex *d_V = NULL; /* right singular vectors */
	int *devInfo = NULL;
	hipfftComplex *d_work = NULL;
	float *d_rwork = NULL;

	int lwork = 0;
	int info_gpu = 0;
	hipsolverDnCreate(&cusolverH);
	
	hipMalloc((void**)&d_A,sizeof(hipfftComplex)*lda*n);
	hipMalloc((void**)&d_S,sizeof(float)*n);
	hipMalloc((void**)&d_U,sizeof(hipfftComplex)*lda*m);
	hipMalloc((void**)&d_V,sizeof(hipfftComplex)*n*n);
	hipMalloc((void**)&devInfo,sizeof(int));

	hipMemcpy(d_A, A, sizeof(hipfftComplex)*lda*n,hipMemcpyHostToDevice); //A传到GPU端
	hipsolverDnCgesvd_bufferSize(
		cusolverH,
		m,
		n,
		&lwork );
	
	hipMalloc((void**)&d_work , sizeof(hipfftComplex)*lwork);

	
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	hipsolverDnCgesvd (
		cusolverH,
		jobu,
		jobvt,
		m,
		n,
		d_A,
		lda,
		d_S,
		d_U,
		lda, // ldu
		d_V,
		n, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);

	hipDeviceSynchronize();

	hipMemcpy(U, d_U, sizeof(hipfftComplex)*lda*m,hipMemcpyDeviceToHost);
	hipMemcpy(V, d_V, sizeof(hipfftComplex)*n*n,hipMemcpyDeviceToHost);
	hipMemcpy(S, d_S, sizeof(float)*n,hipMemcpyDeviceToHost);
	hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	

	//printf("after gesvd: info_gpu = %d\n", info_gpu);

	//printf("=====\n");

	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_V);
	hipFree(devInfo);
	hipFree(d_work);
	hipFree(d_rwork);
	
	hipsolverDnDestroy(cusolverH);
	hipDeviceReset();

}

void cuinverse(hipfftComplex *A,hipfftComplex *A_f,int m){  //A 为原矩阵，A_f为逆矩阵
	hipfftComplex *U = new hipfftComplex[m*m];  //存放左特征向量
	hipfftComplex *V = new hipfftComplex[m*m];	//存放右特征向量
	float *S = new float[m];
	hipfftComplex *UT = new hipfftComplex[m*m];
	hipfftComplex *VT = new hipfftComplex[m*m];

	Mfsvd(A,U,S,V,m,m);  //实现方矩阵的奇异值分解

	/*for(int i = 0;i<m;i++){
		cout<<S[i]<<" "<<endl;
	}
	cout<<endl;
	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<U[j*m+i].x<<"+"<<U[j*m+i].y<<"i"<<" ";
		}
		cout<<endl;
	}
	cout<<"_____+++____"<<endl;

	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<V[j*m+i].x<<"+"<<V[j*m+i].y<<"i"<<" ";
		}
		cout<<endl;
	}                        //V里存的是V的转置
	cout<<"_____"<<endl;
*/
	
	for(int i = 0;i<m;i++){
		S[i] = 1/S[i];
		//cout<<S[i]<<" "<<endl;
	}
	cout<<endl;      //S阵取逆就是取倒数

	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			VT[i*m+j].x= V[j*m+i].x;
			VT[i*m+j].y= 0-V[j*m+i].y;
			
		}
	}   
 	/*for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<VT[j*m+i].x<<"+"<<VT[j*m+i].y<<"i"<<" ";
		}
	cout<<endl;
	}
	cout<<" +++"<<endl;
*/
	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			VT[i*m+j].x = (VT[i*m+j].x)*S[i];
			VT[i*m+j].y = (VT[i*m+j].y)*S[i];
		}
	}
	
	/* for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<VT[j*m+i].x<<"+"<<VT[j*m+i].y<<"i"<<" ";
		}
	cout<<endl;
	}
*/
	 for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			UT[i*m+j].x = U[j*m+i].x;  //U的逆就是转置
			UT[i*m+j].y = 0 - U[j*m+i].y;
		}
		
	}
	

	/*for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<UT[j*m+i].x<<"+"<<UT[j*m+i].y<<"i"<<" ";
		}
		cout<<endl;
	}                        //U的转置UT
	cout<<"_____"<<endl;
	*/

	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			U[j*m+i] = UT[i*m+j];
			
		}
	}          //将UT按行存储

	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			V[j*m+i] = VT[i*m+j];
			
		}  //将VT按行存储
	} 

	/*for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<V[i*m+j].x<<"+"<<V[i*m+j].y<<"i"<<" ";
		}
		cout<<endl;
	}                        //V
	cout<<"_____"<<endl;
	for(int i = 0;i<m;i++){
		for(int j = 0;j<m;j++){
			cout<<U[i*m+j].x<<"+"<<U[i*m+j].y<<"i"<<" ";
		}
		cout<<endl;
	}                        //U
	cout<<"_____"<<endl;
*/
	//printfTensor(m,m,1,V);
	//printfTensor(m,m,1,U);
	
	mul_pro(V,U,A_f,m,m,m);//A_f为逆矩阵
	//printfTensor(m,m,1,A_f);

	
}

float psnr(float *image1,float *image2,int m,int n,int k){

	float PSNR = 0.0;
	float MSE = 0.0;

	for(int i = 0;i<m*n*k;i++){
		image1[i] = image1[i]*255;
		image2[i] = image2[i]*255;
	}
	
	for(int j = 0;j<k;j++){
		for(int a = 0;a<m*n;a++){
			MSE = MSE+((image1[a] - image2[a])*(image1[a] - image2[a]));		
		}	
		MSE = MSE/m*n;
		PSNR = PSNR+10*log10(255*255/MSE);

	}
	PSNR = PSNR/k;
	return PSNR;

}












