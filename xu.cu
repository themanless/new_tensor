#include<stdio.h>
#include<stdlib.h>
#include<hipsolver.h>
#include<hipsparse.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
/**
 * @ complex tensor: a,b,c
 * Create on:May 9 2018
 * @author:da xu
 *
 */
void t(const int m,const int n,const int k,hipComplex* Aarray[],
		hipComplex* Barray[],hipComplex* Carray[],int batchCount){
	hipblasOperation_t transa=HIPBLAS_OP_N;
	hipblasOperation_t transb=HIPBLAS_OP_N;
	hipComplex alpha;
	alpha.x=1;
	alpha.y=0;
	hipComplex beta;
	beta.x=0;
	beta.y=0;
	int lda=m;
	int ldb=k;
	int ldc=m;
    hipComplex *d_Aarray[batchCount];
    hipComplex *d_Barray[batchCount];
	hipComplex *d_Carray[batchCount];
	for(int i=0;i<batchCount;i++){
	hipMalloc((void**)&d_Aarray[i],sizeof(hipComplex)*m*k);
	hipMalloc((void**)&d_Barray[i],sizeof(hipComplex)*k*n);
	hipMalloc((void**)&d_Carray[i],sizeof(hipComplex)*m*n);
	hipMemcpy(d_Aarray[i],Aarray[i],sizeof(hipComplex)*m*k,hipMemcpyHostToDevice);
	hipMemcpy(d_Barray[i],Barray[i],sizeof(hipComplex)*k*n,hipMemcpyHostToDevice);
	}
    const hipComplex **d_A;
    const hipComplex **d_B;
    hipComplex **d_C;
    hipMalloc((void**)&d_A, sizeof(hipComplex *)*batchCount);
    hipMalloc((void**)&d_B, sizeof(hipComplex *)*batchCount);
    hipMalloc((void**)&d_C, sizeof(hipComplex *)*batchCount);
	hipMemcpy(d_A, d_Aarray, sizeof(hipComplex *)*batchCount, hipMemcpyHostToDevice);
	hipMemcpy(d_B, d_Barray, sizeof(hipComplex *)*batchCount, hipMemcpyHostToDevice);
	hipMemcpy(d_C, d_Carray, sizeof(hipComplex *)*batchCount, hipMemcpyHostToDevice);
	//const cuComplex* a[batchCount]={d_Aarray[0],d_Aarray[1],d_Aarray[2],d_Aarray[3]};
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	stat=hipblasCgemmBatched(handle,
			transa,
			transb,
			m,n,k,
			&alpha,
			d_A,lda,
			d_B,ldb,
			&beta,
			d_C,ldc,
			batchCount);
	if(stat==HIPBLAS_STATUS_SUCCESS){
		printf("success\n");
	}
	for(int i=0;i<batchCount;i++){
	hipMemcpy(Carray[i],d_Carray[i],sizeof(hipComplex)*m*n,hipMemcpyDeviceToHost);
	}
	hipblasDestroy(handle);
    for (int i=0; i<batchCount; i++)
    {
        hipFree(d_Aarray[i]);
        hipFree(d_Barray[i]);
        hipFree(d_Carray[i]);
    }
    hipFree(d_C);
    hipFree(d_A);
    hipFree(d_B);


}
int main(int arc ,char** argv){
	/*const int m=4;
	const int n=2;
	const int nnz=4;
	printf("kds");
	int row[5]={0,1,2,3,4};
	int col[4]={0,0,1,1};
	double A[4]={1.0,2.0,3.0,4.0};
	cusparseHandle_t handle1;
	cusparseCreate(&handle1);
	cusparseMatDescr_t descrA;
	cusparseCreateMatDescr(&descrA);
	cusparseSetMatType(descrA, CUSPARSE_MATRIX_TYPE_GENERAL);
	cusparseSetMatIndexBase(descrA, CUSPARSE_INDEX_BASE_ZERO);
	double* b=(double*)malloc(sizeof(double)*m);
	b[0]=1.0; b[1]=2.0;b[2]=4.0;b[3]=5.0;
	double tol=0.000001;
	int rank;
	double* x=(double*)malloc(sizeof(double)*n);
	int* p=(int*)malloc(sizeof(int)*4);
	double  min_norm;
	cusolverSpHandle_t handle;
	cusolverSpCreate(&handle);
	cusolverSpDcsrlsqvqrHost(handle,m,n,nnz,descrA,
			A,row,col,b,
			0.000001,&rank,x,p,&min_norm);
	for(int i=0;i<n;i++){
		printf("X[%d]=%f\n",i,x[i]);
	}*/

	int m=2;int n=2; int k=2;int batchCount=4;
	hipComplex *Aarray[batchCount];
	hipComplex* Barray[batchCount];
	hipComplex* Carray[batchCount];
	for(int i=0;i<batchCount;i++){
    Aarray[i]=(hipComplex*)malloc(sizeof(hipComplex)*m*k);
    Barray[i]=(hipComplex*)malloc(sizeof(hipComplex)*k*n);
    Carray[i]=(hipComplex*)malloc(sizeof(hipComplex)*m*n*2);

	}
   for(int i=0;i<batchCount;i++){
	   for(int j=0;j<m*k;j++){
	   Aarray[i][j].x=j;
	   Aarray[i][j].y=0;}
   }
   for(int i=0;i<batchCount;i++){
   	   for(int j=0;j<n*k;j++){
   	   Barray[i][j].x=j+1;
   	   Barray[i][j].y=0;
      }
   }
   t(m,n,k,Aarray,Barray,Carray,batchCount);
for(int i=0;i<batchCount;i++){
	for(int j=0;j<m*n;j++){
	printf("%f %f       ",Carray[i][j].x,Carray[i][j].y);}

	printf("\n##########################################\n");

}
	return 0;
}
